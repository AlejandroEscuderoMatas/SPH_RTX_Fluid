#include "hip/hip_runtime.h"
#pragma once

#include "SPHCuda.h"
#include <stdio.h>

__device__ int3 calculate_cell_idx(float3 pos_float3, float3 min_box_bound, float3 cell_dims, float h_inv, size_t tid) {

    float3 v = (pos_float3 - (min_box_bound)) * h_inv;
    int3 cell_idx = make_int3(int(v.x - (v.x - floorf(v.x))), int(v.y - (v.y - floorf(v.y))), int(v.z - (v.z - floorf(v.z)))); v;

    if (cell_idx.x < 0 || cell_idx.y < 0 || cell_idx.z < 0 || cell_idx.x >= cell_dims.x || cell_idx.y >= cell_dims.y || cell_idx.z >= cell_dims.z) {
        // Avoid illegal memory access
        return make_int3(-1, -1, -1);
    }

    return cell_idx;

}

/* --------------------- KERNEL FUNCs ----------------------*/

__global__ void updateParticlesDensityAndPressureKernel(
    int numParticles,
    double hsq,
    double partMass,
    double poly6K,
    double initialDensity,
    double pressureK,
    glm::vec3* pos_list,
    double* density_list,
    double* pressure_list,
    int* neighboursListData,
    int* neighboursListOffsets
) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numParticles) {
        int pi = tid;
        double density = 0.0;
        glm::vec3 r;

        // Iterate through neighbors using the flattened list and offsets
        int start = neighboursListOffsets[tid];
        int end = neighboursListOffsets[tid + 1];

        for (int i = start; i < end; ++i) {
            int pj = neighboursListData[i];
            r = pos_list[pi] - pos_list[pj];
            const double rsq = glm::dot(r, r);
            density += partMass * poly6K * glm::pow(hsq - rsq, 3);
        }

        density_list[pi] = fmax(density, initialDensity);
        pressure_list[pi] = pressureK * (density_list[pi] - initialDensity);
    }
}

__global__ void updateParticlesAccelerationKernel(
    int numParticles,
    double h,
    double spikeyK,
    double viscoK,
    double maxAcc,
    glm::vec3* pos_list,
    glm::vec3* vel_list,
    double* density_list,
    double* pressure_list,
    glm::vec3* acc_list,
    int* neighboursListData,
    int* neighboursListOffsets
) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numParticles) {
        glm::vec3 acc = glm::vec3(0.0, 0.0, 0.0);
        glm::vec3 r;
        glm::vec3 vdiff;
        int pi = tid;
        // Iterate through neighbors using the flattened list and offsets
        int start = neighboursListOffsets[tid];
        int end = neighboursListOffsets[tid + 1];

        for (int i = start; i < end; ++i) {
            int pj = neighboursListData[i];
            r = pos_list[pi] - pos_list[pj];
            double dist = glm::length(r);

            if (dist == 0.0) {
                continue;
            }

            float inv = 1 / dist;
            r = inv * r;

            // acceleration due to pressure
            float diff = h - dist;
            float spikey = spikeyK * diff * diff;
            float massRatio = 1; // mj = mi, mj/mi = 1
            float pterm = (pressure_list[pi] + pressure_list[pj]) / (2 * density_list[pi] * density_list[pj]);
            acc -= (float)(massRatio * pterm * spikey) * r;

            // acceleration due to viscosity
            float lap = viscoK * diff;
            vdiff = vel_list[pj] - vel_list[pi];
            acc += (float)(viscoK * massRatio * (1 / density_list[pj]) * lap) * vdiff;
        }

        // acceleration due to gravity
        acc += glm::vec3(0.0, -9.8, 0.0);  // Assuming gravity along the y-axis

        double acc_length = glm::length(acc);

        if (acc_length > maxAcc) {
            acc = (acc / (float)acc_length) * (float)maxAcc;
        }

        acc_list[pi] = acc;
    }
}

__global__ void updateParticlesPositionAndBoundsKernel(
    int numParticles,
    glm::vec3* pos_list,
    glm::vec3* vel_list,
    glm::vec3* acc_list,
    double deltaTime,
    double maxVel,
    double xLimitMin,
    double xLimitMax,
    double yLimitMin,
    double yLimitMax,
    double zLimitMin,
    double zLimitMax,
    double eps,
    double damping

) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numParticles)
    {
        // Update velocity and position
        vel_list[tid] += ((float)deltaTime * acc_list[tid]);
        pos_list[tid] += ((float)deltaTime * vel_list[tid]);

        // Check if velocity exceeds the maximum
        if (glm::length(vel_list[tid]) > maxVel)
        {
            vel_list[tid] = (float)maxVel * glm::normalize(vel_list[tid]);
        }

        // Handle special case when idx is 0
        if (tid == 0)
        {
            // Do something specific for idx == 0
            printf("Y_POS_PART01: %f\n", pos_list[tid].y);
            printf("Y_VEL_PART01: %f\n", vel_list[tid].y);
        }

        // Update plane bounds
        if (pos_list[tid].x > xLimitMax) {
            pos_list[tid].x = (float)(xLimitMax - eps);
            vel_list[tid].x = (float)(-damping * vel_list[tid].x);
        }
        else if (pos_list[tid].x < xLimitMin) {
            pos_list[tid].x = (float)(xLimitMin + eps);
            vel_list[tid].x = (float)(-damping * vel_list[tid].x);
        }

        if (pos_list[tid].z > zLimitMax) {
            pos_list[tid].z = (float)(zLimitMax - eps);
            vel_list[tid].z = (float)(-damping * vel_list[tid].z);
        }
        else if (pos_list[tid].z < zLimitMin) {
            pos_list[tid].z = (float)(zLimitMin + eps);
            vel_list[tid].z = (float)(-damping * vel_list[tid].z);
        }

        if (pos_list[tid].y > yLimitMax) {
            pos_list[tid].y = (float)(yLimitMax - eps);
            vel_list[tid].y = (float)(-damping * vel_list[tid].y);
        }
        else if (pos_list[tid].y < yLimitMin) {
            pos_list[tid].y = (float)(yLimitMin + eps);
            vel_list[tid].y = (float)(-damping * vel_list[tid].y);
        }
    }
}

__global__ void resetCellGridKernel(int* d_cell_list, int numParticles)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (tid < numParticles) {
        d_cell_list[tid] = -1;
    }
}

__global__ void assingCellIdKernel(
    glm::vec3* pos_list,
    int* cell_list,
    int* particle_list,
    int N,
    float3 cell_dims,
    float3 min_box_bound,
    float h_inv
)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (tid <= N) {
        float3 pos_float3;
        pos_float3.x = pos_list[tid].x;
        pos_float3.y = pos_list[tid].y;
        pos_float3.z = pos_list[tid].z;
        int3 cell_idx = calculate_cell_idx(pos_float3, min_box_bound, cell_dims, h_inv, tid);

        if (cell_idx.x != -1) {
            int flat_cell_idx = cell_idx.x + cell_dims.x * cell_idx.y + cell_dims.x * cell_dims.y * cell_idx.z;

            particle_list[tid] = atomicExch(&cell_list[flat_cell_idx], tid);

        }
    }
}

__global__ void densityAndPressureKernel(
    glm::vec3* pos_list,
    int* cell_list,
    int* particle_list,
    double* density_list,
    float3 cell_dims,
    float3 min_box_bound,
    int numParticles,
    double hsq,
    float h_inv,
    double partMass,
    double poly6K,
    double initialDensity,
    double pressureK,
    double* pressure_list
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numParticles)
    {
        int pi = tid;
        float3 pos_float3;

        pos_float3.x = pos_list[tid].x;
        pos_float3.y = pos_list[tid].y;
        pos_float3.z = pos_list[tid].z;

        int3 cell_idx = calculate_cell_idx(pos_float3, min_box_bound, cell_dims, h_inv, tid);

        if (cell_idx.x == -1) {
            return;
        }

        double density = 0.0;
        glm::vec3 r;

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++)
                {
                    int3 neighbor_cell_idx = cell_idx + make_int3(x, y, z);
                    if (neighbor_cell_idx.x < 0 || neighbor_cell_idx.y < 0 || neighbor_cell_idx.z < 0 || neighbor_cell_idx.x >= cell_dims.x || neighbor_cell_idx.y >= cell_dims.y || neighbor_cell_idx.z >= cell_dims.z) {
                        continue;
                    }
                    int neighbor_flat_idx = neighbor_cell_idx.x + neighbor_cell_idx.y * cell_dims.x + neighbor_cell_idx.z * cell_dims.x * cell_dims.y;

                    int neighbor_particle_idx = cell_list[neighbor_flat_idx];
                    while (neighbor_particle_idx != -1)
                    {
                        int pj = neighbor_particle_idx;
                        r = pos_list[pi] - pos_list[pj];
                        const double rsq = glm::dot(r, r);
                        density += partMass * poly6K * glm::pow(hsq - rsq, 3);
                        neighbor_particle_idx = particle_list[neighbor_particle_idx];
                    }
                }
            }
        }
        density_list[pi] = fmax(density, initialDensity);
        pressure_list[pi] = pressureK * (density_list[pi] - initialDensity);
    }
}

__global__ void accelerationKernel(
    glm::vec3* pos_list,
    int* cell_list,
    int* particle_list,
    double* density_list,
    float3 cell_dims,
    float3 min_box_bound,
    int numParticles,
    double h,
    float h_inv,
    double* pressure_list,
    double spikeyK,
    double viscoK,
    double maxAcc,
    glm::vec3* vel_list,
    glm::vec3* acc_list
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numParticles)
    {
        int pi = tid;
        float3 pos_float3_pi;
        glm::vec3 acc = glm::vec3(0.0, 0.0, 0.0);
        glm::vec3 r;
        glm::vec3 vdiff;

        pos_float3_pi.x = pos_list[tid].x;
        pos_float3_pi.y = pos_list[tid].y;
        pos_float3_pi.z = pos_list[tid].z;

        int3 cell_idx = calculate_cell_idx(pos_float3_pi, min_box_bound, cell_dims, h_inv, tid);

        if (cell_idx.x == -1) {
            return;
        }

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++)
                {
                    int3 neighbor_cell_idx = cell_idx + make_int3(x, y, z);
                    if (neighbor_cell_idx.x < 0 || neighbor_cell_idx.y < 0 || neighbor_cell_idx.z < 0 || neighbor_cell_idx.x >= cell_dims.x || neighbor_cell_idx.y >= cell_dims.y || neighbor_cell_idx.z >= cell_dims.z) {
                        continue;
                    }
                    int neighbor_flat_idx = neighbor_cell_idx.x + neighbor_cell_idx.y * cell_dims.x + neighbor_cell_idx.z * cell_dims.x * cell_dims.y;
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx];

                    while (neighbor_particle_idx != -1)
                    {
                        int pj = neighbor_particle_idx;

                        r = pos_list[pi] - pos_list[pj];
                        double dist = length(r);

                        if (dist > 0 && dist < h) {
                            float inv = 1 / dist;
                            r = inv * r;

                            // acceleration due to pressure
                            float diff = h - dist;
                            float spikey = spikeyK * diff * diff;
                            float massRatio = 1; // mj = mi, mj/mi = 1
                            float pterm = (pressure_list[pi] + pressure_list[pj]) / (2 * density_list[pi] * density_list[pj]);
                            acc -= (float)(massRatio * pterm * spikey) * r;

                            // acceleration due to viscosity
                            float lap = viscoK * diff;
                            vdiff = vel_list[pj] - vel_list[pi];
                            acc += (float)(viscoK * massRatio * (1 / density_list[pj]) * lap) * vdiff;
                        }

                        neighbor_particle_idx = particle_list[neighbor_particle_idx];
                    }
                }
            }
        }

        // acceleration due to gravity
        acc += glm::vec3(0.0, -9.8, 0.0);  // Assuming gravity along the y-axis

        double acc_length = glm::length(acc);

        if (acc_length > maxAcc) {
            acc = (acc / (float)acc_length) * (float)maxAcc;
        }

        acc_list[pi] = acc;
    }
}


/* --------------------- AUX FUNC ---------------------- */
void updateParticlesDensityAndPressureCUDA(
    int numParticles,
    double hsq,
    double partMass,
    double poly6K,
    double initialDensity,
    double pressureK,
    glm::vec3* pos_list,
    double* density_list,
    double* pressure_list,
    int* neighboursListData,
    int* neighboursListOffsets
) {

    int numBlocks = (numParticles + 255) / 256;
    int numThreads = 256;

    updateParticlesDensityAndPressureKernel <<< numBlocks, numThreads >>> (
        numParticles,
        hsq,
        partMass,
        poly6K,
        initialDensity,
        pressureK,
        pos_list,
        density_list,
        pressure_list,
        neighboursListData,
        neighboursListOffsets
        );
}

void updateParticlesAccelerationCUDA(
    int numParticles,
    double h,
    double spikeyK,
    double viscoK,
    double maxAcc,
    glm::vec3* pos_list,
    glm::vec3* vel_list,
    double* density_list,
    double* pressure_list,
    glm::vec3* acc_list,
    int* neighboursListData,
    int* neighboursOffsets
) {

    int numBlocks = (numParticles + 255) / 256;
    int numThreads = 256;

    updateParticlesAccelerationKernel << <numBlocks, numThreads >> > (
        numParticles,
        h,
        spikeyK,
        viscoK,
        maxAcc,
        pos_list,
        vel_list,
        density_list,
        pressure_list,
        acc_list,
        neighboursListData,
        neighboursOffsets
        );
}


void updateParticlesPositionAndBoundsCUDA(
    int numParticles,
    glm::vec3* pos_list,
    glm::vec3* vel_list,
    glm::vec3* acc_list,
    double deltaTime,
    double maxVel,
    double xLimitMin,
    double xLimitMax,
    double yLimitMin,
    double yLimitMax,
    double zLimitMin,
    double zLimitMax,
    double eps,
    double damping,
    int threadGroupsPart,
    int threadsPerGroup
)
{
    int numBlocks = (numParticles + 255) / 256;
    int numThreads = 256;

    updateParticlesPositionAndBoundsKernel <<<threadGroupsPart, threadsPerGroup>>> (
        numParticles,
        pos_list,
        vel_list,
        acc_list,
        deltaTime,
        maxVel,
        xLimitMin,
        xLimitMax,
        yLimitMin,
        yLimitMax,
        zLimitMin,
        zLimitMax,
        eps,
        damping
        );
}

void resetCellGridCUDA(
    int* d_cell_list,
    int numParticles,
    int threadGroupsCell,
    int threadsPerGroup
)
{
    resetCellGridKernel <<<threadGroupsCell, threadsPerGroup>>> (d_cell_list, numParticles);
}

void assingCellIdCUDA(
    glm::vec3* pos_list,
    int* cell_list,
    int* particle_list,
    int N,
    float3 cell_dims,
    float3 min_box_bound,
    float h_inv,
    int threadGroupsPart,
    int threadsPerGroup
)
{
    assingCellIdKernel <<<threadGroupsPart, threadsPerGroup>>> (
        pos_list,
        cell_list,
        particle_list,
        N,
        cell_dims,
        min_box_bound,
        h_inv
        );
}

void densityAndPressureCUDA(
    glm::vec3* pos_list,
    int* cell_list,
    int* particle_list,
    double* density_list,
    float3 cell_dims,
    float3 min_box_bound,
    int numParticles,
    double hsq,
    float h_inv,
    double partMass,
    double poly6K,
    double initialDensity,
    double pressureK,
    double* pressure_list,
    int threadGroupsPart,
    int threadsPerGroup
)
{
    densityAndPressureKernel <<<threadGroupsPart, threadsPerGroup>>> (
        pos_list,
        cell_list,
        particle_list,
        density_list,
        cell_dims,
        min_box_bound,
        numParticles,
        hsq,
        h_inv,
        partMass,
        poly6K,
        initialDensity,
        pressureK,
        pressure_list
        );
}

void accelerationCUDA(
    glm::vec3* pos_list,
    int* cell_list,
    int* particle_list,
    double* density_list,
    float3 cell_dims,
    float3 min_box_bound,
    int numParticles,
    double h,
    float h_inv,
    double* pressure_list,
    double spikeyK,
    double viscoK,
    double maxAcc,
    glm::vec3* vel_list,
    glm::vec3* acc_list,
    int threadGroupsPart,
    int threadsPerGroup
)
{
    accelerationKernel <<<threadGroupsPart, threadsPerGroup>>> (
        pos_list,
        cell_list,
        particle_list,
        density_list,
        cell_dims,
        min_box_bound,
        numParticles,
        h,
        h_inv,
        pressure_list,
        spikeyK,
        viscoK,
        maxAcc,
        vel_list,
        acc_list
        );
}
